#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

void gpu_init();
int gcd(int a, int b);

// updated message the gpu_init() function
int clock_speed;
int number_multi_processors;
int number_blocks;
int number_threads;
int max_threads_per_mp;

int num_messages;

hipEvent_t start, stop;

#define ROTL64(x, y) (((x) << (y)) | ((x) >> (64 - (y))))

__device__ const char *chars = " !\"#$%&\'()*+'-./0123456789:;<=>?@ABCDEFGHIJKLMOPQRSTUVWXYZ[\\]^_`abcdefghijklmnopqrstuvwxyz{|}~";
__device__ const uint64_t RC[24] = { 0x0000000000000001, 0x0000000000008082, 0x800000000000808a, 0x8000000080008000, 0x000000000000808b, 0x0000000080000001, 0x8000000080008081, 0x8000000000008009, 0x000000000000008a, 0x0000000000000088, 0x0000000080008009, 0x000000008000000a, 0x000000008000808b, 0x800000000000008b, 0x8000000000008089, 0x8000000000008003, 0x8000000000008002, 0x8000000000000080, 0x000000000000800a, 0x800000008000000a, 0x8000000080008081, 0x8000000000008080, 0x0000000080000001, 0x8000000080008008 };
__device__ const int r[24] = { 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 2, 14, 27, 41, 56, 8, 25, 43, 62, 18, 39, 61, 20, 44 };
__device__ const int piln[24] = { 10, 7, 11, 17, 18, 3, 5, 16, 8, 21, 24, 4, 15, 23, 19, 13, 12, 2, 20, 14, 22, 9, 6, 1 };

__device__ void keccak256(uint64_t state[25])
{
    uint64_t temp, C[5];
	int j;

    for (int i = 0; i < 24; i++) {
        // Theta
		// for i = 0 to 5
		//    C[i] = state[i] ^ state[i + 5] ^ state[i + 10] ^ state[i + 15] ^ state[i + 20];
		C[0] = state[0] ^ state[5] ^ state[10] ^ state[15] ^ state[20];
		C[1] = state[1] ^ state[6] ^ state[11] ^ state[16] ^ state[21];
		C[2] = state[2] ^ state[7] ^ state[12] ^ state[17] ^ state[22];
		C[3] = state[3] ^ state[8] ^ state[13] ^ state[18] ^ state[23];
		C[4] = state[4] ^ state[9] ^ state[14] ^ state[19] ^ state[24];

		// for i = 0 to 5
		//     temp = C[(i + 4) % 5] ^ ROTL64(C[(i + 1) % 5], 1);
		//     for j = 0 to 25, j += 5
		//          state[j + i] ^= temp;
		temp = C[4] ^ ROTL64(C[1], 1); state[0] ^= temp; state[5] ^= temp; state[10] ^= temp; state[15] ^= temp; state[20] ^= temp;
		temp = C[0] ^ ROTL64(C[2], 1); state[1] ^= temp; state[6] ^= temp; state[11] ^= temp; state[16] ^= temp; state[21] ^= temp;
		temp = C[1] ^ ROTL64(C[3], 1); state[2] ^= temp; state[7] ^= temp; state[12] ^= temp; state[17] ^= temp; state[22] ^= temp;
		temp = C[2] ^ ROTL64(C[4], 1); state[3] ^= temp; state[8] ^= temp; state[13] ^= temp; state[18] ^= temp; state[23] ^= temp;
		temp = C[3] ^ ROTL64(C[0], 1); state[4] ^= temp; state[9] ^= temp; state[14] ^= temp; state[19] ^= temp; state[24] ^= temp;

        // Rho Pi
		// for i = 0 to 24
		//     j = piln[i];
		//     C[0] = state[j];
		//     state[j] = ROTL64(temp, r[i]);
		//     temp = C[0];
		temp = state[1];
		j = piln[0]; C[0] = state[j]; state[j] = ROTL64(temp, r[0]); temp = C[0];
		j = piln[1]; C[0] = state[j]; state[j] = ROTL64(temp, r[1]); temp = C[0];
		j = piln[2]; C[0] = state[j]; state[j] = ROTL64(temp, r[2]); temp = C[0];
		j = piln[3]; C[0] = state[j]; state[j] = ROTL64(temp, r[3]); temp = C[0];
		j = piln[4]; C[0] = state[j]; state[j] = ROTL64(temp, r[4]); temp = C[0];
		j = piln[5]; C[0] = state[j]; state[j] = ROTL64(temp, r[5]); temp = C[0];
		j = piln[6]; C[0] = state[j]; state[j] = ROTL64(temp, r[6]); temp = C[0];
		j = piln[7]; C[0] = state[j]; state[j] = ROTL64(temp, r[7]); temp = C[0];
		j = piln[8]; C[0] = state[j]; state[j] = ROTL64(temp, r[8]); temp = C[0];
		j = piln[9]; C[0] = state[j]; state[j] = ROTL64(temp, r[9]); temp = C[0];
		j = piln[10]; C[0] = state[j]; state[j] = ROTL64(temp, r[10]); temp = C[0];
		j = piln[11]; C[0] = state[j]; state[j] = ROTL64(temp, r[11]); temp = C[0];
		j = piln[12]; C[0] = state[j]; state[j] = ROTL64(temp, r[12]); temp = C[0];
		j = piln[13]; C[0] = state[j]; state[j] = ROTL64(temp, r[13]); temp = C[0];
		j = piln[14]; C[0] = state[j]; state[j] = ROTL64(temp, r[14]); temp = C[0];
		j = piln[15]; C[0] = state[j]; state[j] = ROTL64(temp, r[15]); temp = C[0];
		j = piln[16]; C[0] = state[j]; state[j] = ROTL64(temp, r[16]); temp = C[0];
		j = piln[17]; C[0] = state[j]; state[j] = ROTL64(temp, r[17]); temp = C[0];
		j = piln[18]; C[0] = state[j]; state[j] = ROTL64(temp, r[18]); temp = C[0];
		j = piln[19]; C[0] = state[j]; state[j] = ROTL64(temp, r[19]); temp = C[0];
		j = piln[20]; C[0] = state[j]; state[j] = ROTL64(temp, r[20]); temp = C[0];
		j = piln[21]; C[0] = state[j]; state[j] = ROTL64(temp, r[21]); temp = C[0];
		j = piln[22]; C[0] = state[j]; state[j] = ROTL64(temp, r[22]); temp = C[0];
		j = piln[23]; C[0] = state[j]; state[j] = ROTL64(temp, r[23]); temp = C[0];

        //  Chi
		// for j = 0 to 25, j += 5
		//     for i = 0 to 5
		//         C[i] = state[j + i];
		//     for i = 0 to 5
		//         state[j + 1] ^= (~C[(i + 1) % 5]) & C[(i + 2) % 5];
		C[0] = state[0]; C[1] = state[1]; C[2] = state[2]; C[3] = state[3]; C[4] = state[4];
		state[0] ^= (~C[1]) & C[2]; state[1] ^= (~C[2]) & C[3]; state[2] ^= (~C[3]) & C[4]; state[3] ^= (~C[4]) & C[0]; state[4] ^= (~C[0]) & C[1];

		C[0] = state[5]; C[1] = state[6]; C[2] = state[7]; C[3] = state[8]; C[4] = state[9];
		state[5] ^= (~C[1]) & C[2]; state[6] ^= (~C[2]) & C[3]; state[7] ^= (~C[3]) & C[4]; state[8] ^= (~C[4]) & C[0]; state[9] ^= (~C[0]) & C[1];

		C[0] = state[10]; C[1] = state[11]; C[2] = state[12]; C[3] = state[13]; C[4] = state[14];
		state[10] ^= (~C[1]) & C[2]; state[11] ^= (~C[2]) & C[3]; state[12] ^= (~C[3]) & C[4]; state[13] ^= (~C[4]) & C[0]; state[14] ^= (~C[0]) & C[1];

		C[0] = state[15]; C[1] = state[16]; C[2] = state[17]; C[3] = state[18]; C[4] = state[19];
		state[15] ^= (~C[1]) & C[2]; state[16] ^= (~C[2]) & C[3]; state[17] ^= (~C[3]) & C[4]; state[18] ^= (~C[4]) & C[0]; state[19] ^= (~C[0]) & C[1];

		C[0] = state[20]; C[1] = state[21]; C[2] = state[22]; C[3] = state[23]; C[4] = state[24];
		state[20] ^= (~C[1]) & C[2]; state[21] ^= (~C[2]) & C[3]; state[22] ^= (~C[3]) & C[4]; state[23] ^= (~C[4]) & C[0]; state[24] ^= (~C[0]) & C[1];

        //  Iota
        state[0] ^= RC[i];
    }
}

__device__ void keccak(const char *message, int message_len, unsigned char *output, int output_len)
{
    uint64_t state[25];
    uint8_t temp[144];
    int rsize = 136;
    int rsize_byte = 17;

    memset(state, 0, sizeof(state));

    for ( ; message_len >= rsize; message_len -= rsize, message += rsize) {
        for (int i = 0; i < rsize_byte; i++) {
            state[i] ^= ((uint64_t *) message)[i];
		}
        keccak256(state);
    }

    // last block and padding
    memcpy(temp, message, message_len);
    temp[message_len++] = 1;
    memset(temp + message_len, 0, rsize - message_len);
    temp[rsize - 1] |= 0x80;

    for (int i = 0; i < rsize_byte; i++) {
        state[i] ^= ((uint64_t *) temp)[i];
	}

    keccak256(state);
    memcpy(output, state, output_len);
}

__device__ void generate_message(uint8_t *message, uint64_t tid)
{
	int len = 0;
	const int num_chars = 94;
    while (len < 8)
	{
		message[len++] = tid % 256;
		tid /= num_chars;
	}
}

__global__ void brute_force_single(uint8_t *d_diff, uint8_t *d_preimage, int *done, uint64_t starting_tid)
{
	const int output_len = 32;
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	unsigned char output[output_len];
	uint8_t current_message[72];
    memcpy(current_message, d_preimage, 64);

	generate_message(current_message + 64, tid + starting_tid);
	keccak((char*)current_message, 72, &output[0], output_len);

    for (int i = 0; i < 32; i++)
    {
        if (output[i] > d_diff[i]) return;
        if (output[i] < d_diff[i]) {
            done[0] = 1;
            memcpy(d_preimage, output, 32);
            memcpy(d_preimage + 32, current_message + 64, 8);
            return;
        }
    }

}

void gpu_init()
{
    hipDeviceProp_t device_prop;
    int block_size;

	hipError_t cudaerr = hipGetDeviceProperties(&device_prop, 0);
    if (cudaerr != hipSuccess) {
		printf("getting properties for device failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        exit(EXIT_FAILURE);
    }

    number_threads = device_prop.maxThreadsPerBlock;
    number_multi_processors = device_prop.multiProcessorCount;
    max_threads_per_mp = device_prop.maxThreadsPerMultiProcessor;
    block_size = (max_threads_per_mp / gcd(max_threads_per_mp, number_threads));
    number_threads = max_threads_per_mp / block_size;
    number_blocks = block_size * number_multi_processors;
    clock_speed = (int) (device_prop.memoryClockRate * 1000 * 1000);    // convert from GHz to hertz
}

int gcd(int a, int b) {
    return (a == 0) ? b : gcd(b % a, a);
}

void find_message()
{
    uint8_t* data = (uint8_t*)malloc(33 * sizeof(uint8_t));
    // read 33 bytes from stdin
	// first byte is reserved for compatibility with the CPU worker
	// rest are the difficulty
    fread(data, 1, 33, stdin);
    uint8_t* diff = data + 1;

	uint64_t starting_tid = 0;

	int *d_done;
	uint8_t *d_diff;
	uint8_t *d_preimage;

	hipMalloc((void**) &d_done, sizeof(int));
	hipMalloc((void**) &d_diff, 32);
	hipMalloc((void**) &d_preimage, 64);
	hipMemcpy(d_diff, diff, 32, hipMemcpyHostToDevice);

	// keep reading proof.hash and pubkey, in total 64 bytes
    while (1) {
        int h_done[1] = {0};
	    hipMemcpy(d_done, h_done, sizeof(int), hipMemcpyHostToDevice);
        uint8_t* preimage = (uint8_t*)malloc(64);
        const size_t ret_code = fread(preimage, 1, 64, stdin);
        if (ret_code != 64) {
            break;
        }

        hipMemcpy(d_preimage, preimage, 64, hipMemcpyHostToDevice);
        int index = 0;
        while (!h_done[0]) {
            index++;
            brute_force_single<<<number_blocks, number_threads>>>(d_diff, d_preimage, d_done, starting_tid);
            starting_tid += number_blocks * number_threads;
            hipMemcpy(h_done, d_done, sizeof(int), hipMemcpyDeviceToHost);
            hipError_t cudaerr = hipDeviceSynchronize();
            if (cudaerr != hipSuccess) {
                h_done[0] = 1;
                printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
            }
        }
        hipMemcpy(preimage, d_preimage, 64, hipMemcpyDeviceToHost);
        fwrite(preimage, 1, 40, stdout);
    }
}

int main(int argc, char **argv)
{
    gpu_init();
	find_message();
    return EXIT_SUCCESS;
}